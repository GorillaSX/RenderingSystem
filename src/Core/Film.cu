#include "hip/hip_runtime.h"
#include "commonHeaders.h"
#include <GL/gl3w.h>

#include <cuda_gl_interop.h>
#include <>
#include "Utils/CudaUtils.h"

#include "Core/Common.h"
#include "App.h"
#include "Core/Film.h"
#include "Tonemappers/Tonemapper.h"
#include "Utils/GLUtils.h"
#include "Utils/SysUtils.h"

using namespace Gorilla;

Film::Film(bool windowed_): windowed(windowed_)
{
}

void Film::initialize()
{
    if(windowed)
    {
        glGenTextures(1, &textureId);

        GLUtils::checkError("Could not create OpenGL texture");
        
        glBindTexture(GL_TEXTURE_2D, textureId);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

        GLUtils::checkError("Could not set OpenGL texture parameters");
    }
}

void Film::shutdown()
{
    if(windowed)
    {
        if(textureResource != nullptr)
        {
            CudaUtils::checkError(hipGraphicsUnregisterResource(textureResource),"Could not unregister OpenGL texture");
            textureResource = nullptr;
        }

        glDeleteTextures(1, &textureId);
        GLUtils::checkError("Could not delete OpenGL texture");
    }
}

void Film::resize(uint32_t width_, uint32_t height_)
{
    width = width_;
    height = height_;
    length = width * height;

    cumulativeImage.resize(width, height);
    normalizedImage.resize(width, height);
    tonemappedImage.resize(width, height);

    if(windowed)
    {
        if(textureResource != nullptr)
        {
            CudaUtils::checkError(hipGraphicsUnregisterResource(textureResource), "Could not unregister OpenGL texture");
            textureResource = nullptr;
        }

        glBindTexture(GL_TEXTURE_2D, textureId);
        glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, GLsizei(width), GLsizei(height), 0, GL_RGBA, GL_FLOAT, nullptr);
        glBindTexture(GL_TEXTURE_2D, 0);

        GLUtils::checkError("Could not reserve OpenGL texture memory");

        CudaUtils::checkError(hipGraphicsGLRegisterImage(&textureResource, textureId, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore), "Could not register OpenGL texture");
    }
    clear();
}

void Film::clear()
{
    cumulativeImage.clear();
    pixelSamples = 0;
    cleared = true;
}

bool Film::hasBeenCleared() const
{
    return cleared;
}

void Film::resetCleared() 
{
    cleared = false;
}

void Film::load(uint32_t width_, uint32_t height_, const std::string& fileName)
{

	uint64_t fileSize = SysUtils::getFileSize(fileName);

	if (fileSize != (width_ * height_ * sizeof(Color)))
		throw std::runtime_error("Film file has wrong size");

	std::ifstream file(fileName, std::ios::in | std::ios::binary);

	if (!file.is_open())
		throw std::runtime_error("Could not open the film file for reading");

	resize(width_, height_);

	file.read(reinterpret_cast<char*>(cumulativeImage.getData()), fileSize);
	file.close();

	cumulativeImage.upload();
}

void Film::loadMultiple(uint32_t width_, uint32_t height_, const std::string& dirName)
{

	resize(width_, height_);

	std::vector<std::string> fileNames = SysUtils::getAllFiles(dirName);
	std::vector<Color> inputData(length);

	Color* inputPtr = inputData.data();
	Color* cumulativePtr = cumulativeImage.getData();

	for (const std::string& fileName : fileNames)
	{
		uint64_t fileSize = SysUtils::getFileSize(fileName);

		if (fileSize != length * sizeof(Color))
			throw std::runtime_error("Film file has wrong size");

		std::ifstream file(fileName, std::ios::in | std::ios::binary);

		if (!file.is_open())
			throw std::runtime_error("Could not open the film file for reading");

		file.read(reinterpret_cast<char*>(inputPtr), fileSize);
		file.close();

		for (uint32_t i = 0; i < length; ++i)
			cumulativePtr[i] += inputPtr[i];
	}

	cumulativeImage.upload();
}

void Film::save(const std::string& fileName)const
{

	std::ofstream file(fileName, std::ios::out | std::ios::binary);

	if (!file.is_open())
		throw std::runtime_error("Could not open the film file for writing");

	file.write(reinterpret_cast<const char*>(cumulativeImage.getData()), sizeof(Color) * length);
	file.close();
}

CUDA_CALLABLE void Film::addSample(uint32_t x, uint32_t y, const Color& color, float filterWeight)
{
	Color temp = cumulativeImage.getPixel(x, y);

	temp.r += color.r * filterWeight;
	temp.g += color.g * filterWeight;
	temp.b += color.b * filterWeight;
	temp.a += filterWeight;

	cumulativeImage.setPixel(x, y, temp);
}

CUDA_CALLABLE void Film::addSample(uint32_t index, const Color& color, float filterWeight)
{
	Color temp = cumulativeImage.getPixel(index);

	temp.r += color.r * filterWeight;
	temp.g += color.g * filterWeight;
	temp.b += color.b * filterWeight;
	temp.a += filterWeight;

	cumulativeImage.setPixel(index, temp);
}

__global__ void normalizeKernel(hipSurfaceObject_t cumulative, hipSurfaceObject_t normalized, uint32_t width, uint32_t height)
{
	uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= width || y >= height)
		return;

	float4 color;
	surf2Dread(&color, cumulative, x * sizeof(float4), y);

	color.x /= color.w;
	color.y /= color.w;
	color.z /= color.w;
	color.w = 1.0f;

	surf2Dwrite(color, normalized, x * sizeof(float4), y);
}


void Film::normalize()
{
		dim3 dimBlock(16, 16);
		dim3 dimGrid;

		dimGrid.x = (width + dimBlock.x - 1) / dimBlock.x;
		dimGrid.y = (height + dimBlock.y - 1) / dimBlock.y;

		normalizeKernel<<<dimGrid, dimBlock>>>(cumulativeImage.getSurfaceObject(), normalizedImage.getSurfaceObject(), width, height);
		CudaUtils::checkError(hipPeekAtLastError(), "Could not launch normalize kernel");
		CudaUtils::checkError(hipDeviceSynchronize(), "Could not execute normalize kernel");
}

__global__ void tonemapKernel(hipSurfaceObject_t input, hipSurfaceObject_t output, uint32_t width, uint32_t height)
{
	uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= width || y >= height)
		return;

	float4 temp;
	surf2Dread(&temp, input, x * sizeof(float4), y);

	Color color(temp.x, temp.y, temp.z, temp.w);
	color.clamp();
	color = Color::pow(color, 1.0f / 2.2f);
	color.a = 1.0f;

	surf2Dwrite(make_float4(color.r, color.g, color.b, color.a), output, x * sizeof(float4), y);
}


void Film::tonemap()
{
		dim3 dimBlock(16, 16);
		dim3 dimGrid;

		dimGrid.x = (width + dimBlock.x - 1) / dimBlock.x;
		dimGrid.y = (height + dimBlock.y - 1) / dimBlock.y;

		tonemapKernel<<<dimGrid, dimBlock>>>(normalizedImage.getSurfaceObject(), tonemappedImage.getSurfaceObject(), width, height);
		CudaUtils::checkError(hipPeekAtLastError(), "Could not launch tonemap kernel");
		CudaUtils::checkError(hipDeviceSynchronize(), "Could not execute tonemap kernel");
}

__global__ void updateTextureKernel(hipSurfaceObject_t input, hipSurfaceObject_t output, uint32_t width, uint32_t height)
{
	uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x >= width || y >= height)
		return;

	float4 color;
	surf2Dread(&color, input, x * sizeof(float4), y);
	surf2Dwrite(color, output, x * sizeof(float4), y);
}

void Film::updateTexture()
{
	if (windowed)
	{
			CudaUtils::checkError(hipGraphicsMapResources(1, &textureResource, 0), "Could not map texture resource");

			hipArray_t textureData;
			CudaUtils::checkError(hipGraphicsSubResourceGetMappedArray(&textureData, textureResource, 0, 0), "Could not get mapped array");

			hipResourceDesc resDesc;
			memset(&resDesc, 0, sizeof(resDesc));
			resDesc.resType = hipResourceTypeArray;
			resDesc.res.array.array = textureData;

			hipSurfaceObject_t surfaceObject;
			CudaUtils::checkError(hipCreateSurfaceObject(&surfaceObject, &resDesc), "Could not create surface object");
			
			dim3 dimBlock(16, 16);
			dim3 dimGrid;

			dimGrid.x = (width + dimBlock.x - 1) / dimBlock.x;
			dimGrid.y = (height + dimBlock.y - 1) / dimBlock.y;

			updateTextureKernel<<<dimGrid, dimBlock>>>(tonemappedImage.getSurfaceObject(), surfaceObject, width, height);
			CudaUtils::checkError(hipPeekAtLastError(), "Could not launch update texture kernel");
			CudaUtils::checkError(hipDeviceSynchronize(), "Could not execute update texture kernel");

			CudaUtils::checkError(hipDestroySurfaceObject(surfaceObject), "Could not destroy surface object");
			CudaUtils::checkError(hipGraphicsUnmapResources(1, &textureResource, 0), "Could not unmap texture resource");
	}
}

Color Film::getCumulativeColor(uint32_t x, uint32_t y) const
{
	return cumulativeImage.getPixel(x, y);
}

Color Film::getNormalizedColor(uint32_t x, uint32_t y) const
{
	return normalizedImage.getPixel(x, y);
}

Color Film::getTonemappedColor(uint32_t x, uint32_t y) const
{
	return tonemappedImage.getPixel(x, y);
}

CUDA_CALLABLE Image& Film::getCumulativeImage()
{
	return cumulativeImage;
}

CUDA_CALLABLE Image& Film::getNormalizedImage()
{
	return normalizedImage;
}

CUDA_CALLABLE Image& Film::getTonemappedImage()
{
	return tonemappedImage;
}

CUDA_CALLABLE uint32_t Film::getWidth() const
{
	return width;
}

CUDA_CALLABLE uint32_t Film::getHeight() const
{
	return height;
}

CUDA_CALLABLE uint32_t Film::getLength() const
{
	return length;
}

GLuint Film::getTextureId() const
{
	return textureId;
}
